#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#pragma pack(1)

typedef struct BITMAPFILEHEADER {
	unsigned short int  bfType;
	unsigned int bfSize;
	unsigned short int  bfReserved1;
	unsigned short int  bfReserved2;
	unsigned int bfOffBits;
} BITMAPFILEHEADER;

typedef struct BITMAPINFOHEADER {
	unsigned int biSize;
	unsigned int biWidth;
	unsigned int biHeight;
	unsigned short int  biPlanes;
	unsigned short int  biBitCount;
	unsigned int biCompression;
	unsigned int biSizeImage;
	unsigned int biXPelsPerMeter;
	unsigned int biYPelsPerMeter;
	unsigned int biClrUsed;
	unsigned int biClrImportant;
} BITMAPINFOHEADER;


static void HandleError(hipError_t err, const char *file, int line){
	if (err != hipSuccess){
		printf("%s in %s at line %d\n",
			hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

//����һ��ͼ��λͼ���ݡ����ߡ���ɫ��ָ�뼰ÿ������ռ��λ������Ϣ,����д��ָ���ļ���
void savebmpfile(char *bmpName, unsigned char *imgBuf, int width, int height, int biBitCount) {
	int colorTablesize = 0;

	if (biBitCount == 8)
		colorTablesize = 1024;  // 8*128

	//���洢ͼ������ÿ���ֽ���Ϊ4�ı���
	int lineByte = (width * biBitCount / 8 + 3) / 4 * 4;
	//�Զ�����д�ķ�ʽ���ļ�
	FILE *fp = fopen(bmpName, "wb");
	//����λͼ�ļ�ͷ�ṹ��������д�ļ�ͷ��Ϣ
	BITMAPFILEHEADER fileHead;

	fileHead.bfType = 0x4D42;  // bmp����
	// bfSize��ͼ���ļ�4����ɲ���֮��
	fileHead.bfSize = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + colorTablesize + lineByte * height;
	fileHead.bfReserved1 = 0;
	fileHead.bfReserved2 = 0;

	// bfOffBits��ͼ���ļ�ǰ3����������ռ�֮��
	fileHead.bfOffBits = 54 + colorTablesize;

	//д�ļ�ͷ���ļ�
	fwrite(&fileHead, sizeof(BITMAPFILEHEADER), 1, fp);

	//����λͼ��Ϣͷ�ṹ��������д��Ϣͷ��Ϣ
	BITMAPINFOHEADER head;

	head.biBitCount = biBitCount;
	head.biClrImportant = 0;
	head.biClrUsed = 0;
	head.biCompression = 0;
	head.biHeight = height;
	head.biPlanes = 1;
	head.biSize = 40;
	head.biSizeImage = lineByte * height;
	head.biWidth = width;
	head.biXPelsPerMeter = 0;
	head.biYPelsPerMeter = 0;

	fwrite(&head, sizeof(BITMAPINFOHEADER), 1, fp);
	fwrite(imgBuf, height * lineByte, 1, fp);

	fclose(fp);
}


__global__ void image_convolution_kernel(unsigned char *pBmpBufGPU, unsigned char *resBufGPU,double *kernelGPU,int bmpWidth, int bmpHeight) {
	double accum;
	int col = threadIdx.x + blockIdx.x * blockDim.x;   //col index
	int row = threadIdx.y + blockIdx.y * blockDim.y;   //row index
	int maskRowsRadius = 2;
	int maskColsRadius = 2;

	for (int k = 0; k < 3; k++) {      //cycle on kernel channels
		if (row < bmpHeight && col < bmpWidth) {
			accum = 0.0;
			int startRow = row - maskRowsRadius;  //row index shifted by mask radius
			int startCol = col - maskColsRadius;  //col index shifted by mask radius
			for (int i = 0; i < 5; i++) { //cycle on mask rows
				for (int j = 0; j < 5; j++) { //cycle on mask columns
					int currentRow = startRow + i; // row index to fetch data from input image
					int currentCol = startCol + j; // col index to fetch data from input image
					if (currentRow >= 0 && currentRow < bmpHeight && currentCol >= 0 && currentCol < bmpWidth) 
						accum += pBmpBufGPU[(currentRow * bmpWidth + currentCol)*3 + k] *kernelGPU[i * 5 + j];
					else 
						accum = 0.0;
				}
			}
			resBufGPU[(row* bmpWidth + col) * 3 + k] = accum;
		}
	}
}

int main() {
	char filename[] = "timg.bmp";
	char writePath[] = "result.bmp";
	int kernelsize = 25;
	const int blocksize=16;
	double kernelx[kernelsize] = { 0.01441881,0.02808402,0.0350727, 0.02808402,0.01441881,
		                           0.02808402,0.05470020,0.06831229,0.05470020,0.02808402,
								   0.03507270,0.06831229,0.08531173,0.06831229,0.03507270,
								   0.02808402,0.05470020,0.06831229,0.05470020,0.02808402,
								   0.01441881,0.02808402,0.03507270,0.02808402,0.01441881};

	unsigned char *resBuf = NULL;
	unsigned char *pBmpBuf = NULL;  //����ͼ�����ݵ�ָ��

	//read bmp
	int bmpWidth;    //ͼ��Ŀ�
	int bmpHeight;   //ͼ��ĸ�
	int BiBitCount;  //ͼ�����ͣ�ÿ����λ�� 8-�Ҷ�ͼ 24-��ɫͼ
	
	BITMAPFILEHEADER BmpHead;
	BITMAPINFOHEADER BmpInfo;

	FILE *fp = fopen(filename, "rb");  //�����ƶ���ʽ��ָ����ͼ���ļ�

	fread(&BmpHead, sizeof(BITMAPFILEHEADER), 1, fp);
	fread(&BmpInfo, sizeof(BITMAPINFOHEADER), 1, fp);

	bmpWidth = BmpInfo.biWidth;
	bmpHeight = BmpInfo.biHeight;
	BiBitCount = BmpInfo.biBitCount;

	int lineByte = (bmpWidth * BiBitCount / 8 + 3) / 4 * 4;

	pBmpBuf = (unsigned char*)malloc(lineByte * bmpHeight);
	resBuf = (unsigned char*)malloc(bmpWidth * 3 * bmpHeight);

	fread(pBmpBuf, lineByte * bmpHeight, 1, fp);
	printf("read bmp file successfully\n");

	//cuda conv
	unsigned char *pBmpBufGPU;
	unsigned char *resBufGPU;
	double *kernelGPU;

	HANDLE_ERROR(hipMalloc(&pBmpBufGPU, 3*bmpWidth * bmpHeight * sizeof(unsigned char)));
	HANDLE_ERROR(hipMalloc(&resBufGPU, 3*bmpWidth * bmpHeight * sizeof(unsigned char)));
	HANDLE_ERROR(hipMalloc(&kernelGPU, kernelsize * sizeof(double)));
	HANDLE_ERROR(hipMemcpy(pBmpBufGPU, pBmpBuf, 3 * bmpWidth * bmpHeight * sizeof(unsigned char), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(kernelGPU, kernelx, kernelsize * sizeof(double), hipMemcpyHostToDevice));

	dim3 dimBlock(blocksize,blocksize,1);
	dim3 dimGrid((bmpWidth+blocksize-1)/blocksize,(bmpHeight+blocksize-1)/blocksize);
	
	printf("cuda global memory convolution\n");
	printf("image dimensions %d %d\n", bmpWidth, bmpHeight);
	image_convolution_kernel <<<dimGrid,dimBlock>>> (pBmpBufGPU, resBufGPU, kernelGPU,bmpWidth, bmpHeight);

	HANDLE_ERROR(hipMemcpy(resBuf, resBufGPU, 3*bmpWidth * bmpHeight * sizeof(unsigned char), hipMemcpyDeviceToHost));
	printf("convolution finished\n");
	
	savebmpfile(writePath, resBuf, bmpWidth, bmpHeight, BiBitCount);
	printf("Save bmp file succussfully\n");
	if (pBmpBuf) free(pBmpBuf);
	if (resBuf) free(resBuf);

	hipFree(pBmpBufGPU);
	hipFree(resBufGPU);

	return 0;
}
